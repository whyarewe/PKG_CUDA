#include "hip/hip_runtime.h"
#include <cstdlib>

#include "CUDAPropagation.h"
#include "CUDAUtils.h"
#include "Config.h"

using namespace CUDAHelpers;

CUDAPropagation::CUDAPropagation(uint32_t x_axis_bound, uint32_t y_axis_bound)
{
	data = nullptr;
	out_data = nullptr;

	VALID(hipMalloc(reinterpret_cast<void**>(&data), x_axis_bound * y_axis_bound * sizeof(float)));
	VALID(hipMalloc(reinterpret_cast<void**>(&out_data), x_axis_bound * y_axis_bound * sizeof(float)));
}

CUDAPropagation::~CUDAPropagation()
{
	hipFree(data);
	hipFree(out_data);
}

auto CUDAPropagation::propagate(const ComputingData& data, const Device device, const Method method) -> void
{
	switch (method)
	{
	case Method::Laplace:
		laplace(data, device);
		break;
	case Method::FTCS:
		ftcs(data, device);
		break;
	case Method::FIS:
		dhe(data, device);
		break;
	default:
		std::cerr << "CUDA Propagation: Critical error, unknown method!" << std::endl;
		std::exit(0);
	}
}

auto CUDAPropagation::laplace(const ComputingData& data, const Device device) -> void
{
	switch (device)
	{
	case Device::CPU:
		laplace_cpu(data.board, data.x_axis_bound, data.y_axis_bound);
		break;
	case Device::GPU:
		laplace_gpu(data.board, data.x_axis_bound, data.y_axis_bound);
	}
}

auto CUDAPropagation::ftcs(const ComputingData& data, const Device device) -> void
{
	switch (device)
	{
	case Device::CPU:
		ftcs_cpu(data.board, data.x_axis_bound, data.y_axis_bound);
		break;
	case Device::GPU:
		ftcs_gpu(data.board, data.x_axis_bound, data.y_axis_bound);
	}
}

auto CUDAPropagation::dhe(const ComputingData& data, const Device device) -> void
{
	switch (device)
	{
	case Device::CPU:
		dhe_cpu(data.board, data.x_axis_bound, data.y_axis_bound);
		break;
	case Device::GPU:
		dhe_gpu(data.board, data.x_axis_bound, data.y_axis_bound);
	}
}


__global__ void kernel_laplace(float* data, float* out_data, const int x_axis_bound, const int y_axis_bound)
{
	const uint16_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	const uint16_t idy = blockIdx.y * blockDim.y + threadIdx.y;

	const auto gid = idy * x_axis_bound + idx;

	if (idx > x_axis_bound || idy > y_axis_bound)
	{
		return;
	}

	out_data[gid] = 0.25f * (data[gid - 1] + data[gid + 1] + data[gid + x_axis_bound] + data[gid - x_axis_bound]);
}

__global__ void kernel_ftcs(float* data, float* out_data, const int x_axis_bound, const int y_axis_bound, const float r, const float r2)
{
	const uint16_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	const uint16_t idy = blockIdx.y * blockDim.y + threadIdx.y;

	const auto gid = idy * x_axis_bound + idx;

	if (idx > x_axis_bound || idy > y_axis_bound)
	{
		return;
	}

	float horizontal = 0.25 * r2 * (data[gid - 1] + data[gid + 1] +
		data[gid + x_axis_bound] + data[gid - x_axis_bound]);

	float diagonal = 0.25 * r * (data[gid - x_axis_bound - 1] + data[gid - x_axis_bound + 1] +
		data[gid + x_axis_bound - 1] + data[gid + x_axis_bound + 1]);

	out_data[gid] = horizontal + diagonal;
}

__global__ void kernel_dhe(float* data, float* out_data, const int x_axis_bound, const int y_axis_bound, const float x_param, const float y_param)
{
	const uint16_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	const uint16_t idy = blockIdx.y * blockDim.y + threadIdx.y;

	const auto gid = idy * x_axis_bound + idx;

	if (idx > x_axis_bound || idy > y_axis_bound)
	{
		return;
	}

	float current_val = data[gid];

	float x_comp = x_param * (data[gid + x_axis_bound] - (2 * current_val) +
		data[gid - x_axis_bound]);

	float y_comp = y_param * (data[gid + 1] - (2 * current_val) +
		data[gid - 1]);

	out_data[gid] = current_val + x_comp + y_comp;
}

auto CUDAPropagation::laplace_cpu(std::vector<float>& vec, const uint32_t x_axis_bound,
	const uint32_t y_axis_bound) -> void
{
	std::vector<float> out_vec(vec.capacity());

	for (auto i = 1u; i < y_axis_bound - 1; ++i)
	{
		for (auto j = 1u; j < x_axis_bound - 1; j++)
		{
			out_vec[i * x_axis_bound + j] = (0.25f * (vec[i * x_axis_bound + j - 1] + vec[i * x_axis_bound + j + 1]
				+ vec[(i + 1) * x_axis_bound + j] + vec[(i - 1) * x_axis_bound + j]));
		}
	}

	vec = out_vec;
}

auto CUDAPropagation::laplace_gpu(std::vector<float>& vec, const uint32_t x_axis_bound,
	const uint32_t y_axis_bound) -> void
{
	VALID(hipMemcpyAsync(data, vec.data(), x_axis_bound * y_axis_bound * sizeof(float), hipMemcpyHostToDevice));

	dim3 block(32, 32);
	dim3 grid(x_axis_bound / block.x, y_axis_bound / block.y);

	kernel_laplace << <grid, block >> > (data, out_data, x_axis_bound, y_axis_bound);

	VALID(hipMemcpyAsync(vec.data(), out_data, x_axis_bound * y_axis_bound * sizeof(float), hipMemcpyDeviceToHost));
}

auto CUDAPropagation::ftcs_cpu(std::vector<float>& vec, const uint32_t x_axis_bound, const uint32_t y_axis_bound) -> void
{
	std::vector<float> out_vec(vec.capacity());

	const float r = (Config::FTCS_Config::alpha * Config::FTCS_Config::dt) / (Config::FTCS_Config::dx * Config::FTCS_Config::dx);
	const float r2 = 1 - 2 * r;

	for (auto i = 1u; i < y_axis_bound - 1; ++i)
	{
		for (auto j = 1u; j < x_axis_bound - 1; j++)
		{
			float horizontal = 0.25 * r2 * (vec[(i - 1) * x_axis_bound + j] + vec[(i + 1) * x_axis_bound + j] +
				vec[i * x_axis_bound + j - 1] + vec[i * x_axis_bound + j + 1]);

			float diagonal = 0.25 * r * (vec[(i - 1) * x_axis_bound + j - 1] + vec[(i - 1) * x_axis_bound + j + 1] +
				vec[(i + 1) * x_axis_bound + j - 1] + vec[(i + 1) * x_axis_bound + j - 1]);

			out_vec[i * x_axis_bound + j] = horizontal + diagonal;
		}
	}

	vec = out_vec;
}

auto CUDAPropagation::ftcs_gpu(std::vector<float>& vec, const uint32_t x_axis_bound,
	const uint32_t y_axis_bound) -> void
{
	const float r = (Config::FTCS_Config::alpha * Config::FTCS_Config::dt) / (Config::FTCS_Config::dx * Config::FTCS_Config::dx);
	const float r2 = 1 - 2 * r;

	VALID(hipMemcpyAsync(data, vec.data(), x_axis_bound * y_axis_bound * sizeof(float), hipMemcpyHostToDevice));

	dim3 block(32, 32);
	dim3 grid(x_axis_bound / block.x, y_axis_bound / block.y);

	kernel_ftcs << <grid, block >> > (data, out_data, x_axis_bound, y_axis_bound, r, r2);

	VALID(hipMemcpyAsync(vec.data(), out_data, x_axis_bound * y_axis_bound * sizeof(float), hipMemcpyDeviceToHost));
}

auto CUDAPropagation::dhe_cpu(std::vector<float>& vec, const uint32_t x_axis_bound, const uint32_t y_axis_bound) -> void
{
	std::vector<float> out_vec(vec.capacity());

	float x_param = ((Config::DHE_Config::K * Config::DHE_Config::dt) /
		(Config::DHE_Config::sh * Config::DHE_Config::density * Config::DHE_Config::dx * Config::DHE_Config::dx));

	float y_param = ((Config::DHE_Config::K * Config::DHE_Config::dt) /
		(Config::DHE_Config::sh * Config::DHE_Config::density * Config::DHE_Config::dy * Config::DHE_Config::dy));

	for (auto i = 1u; i < y_axis_bound - 1; ++i)
	{
		for (auto j = 1u; j < x_axis_bound - 1; j++)
		{
			float current_val = vec[i * x_axis_bound + j];

			float x_comp = x_param * (vec[(i + 1) * x_axis_bound + j] - (2 * current_val) +
				vec[(i - 1) * x_axis_bound + j]);

			float y_comp = y_param * (vec[i * x_axis_bound + j + 1] - (2 * current_val) +
				vec[i * x_axis_bound + j - 1]);

			out_vec[i * x_axis_bound + j] = current_val + x_comp + y_comp;
		}
	}

	vec = out_vec;
}

auto CUDAPropagation::dhe_gpu(std::vector<float>& vec, const uint32_t x_axis_bound,
	const uint32_t y_axis_bound) -> void
{
	float x_param = ((Config::DHE_Config::K * Config::DHE_Config::dt) /
		(Config::DHE_Config::sh * Config::DHE_Config::density * Config::DHE_Config::dx * Config::DHE_Config::dx));

	float y_param = ((Config::DHE_Config::K * Config::DHE_Config::dt) /
		(Config::DHE_Config::sh * Config::DHE_Config::density * Config::DHE_Config::dy * Config::DHE_Config::dy));

	VALID(hipMemcpyAsync(data, vec.data(), x_axis_bound * y_axis_bound * sizeof(float), hipMemcpyHostToDevice));

	dim3 block(32, 32);
	dim3 grid(x_axis_bound / block.x, y_axis_bound / block.y);

	kernel_dhe << <grid, block >> > (data, out_data, x_axis_bound, y_axis_bound, x_param, y_param);

	VALID(hipMemcpyAsync(vec.data(), out_data, x_axis_bound * y_axis_bound * sizeof(float), hipMemcpyDeviceToHost));
}